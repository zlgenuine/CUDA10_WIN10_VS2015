#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <time.h>
#include <iostream>

using namespace std;

//define a Matrix's dimensionality

int const M = 5;
int const N = 10;

int main()
{
	//declare a state variable 

	hipblasStatus_t status;

	//assign space for defined matrix
	float *h_A = (float*)malloc(N*M * sizeof(float));
	float *h_B = (float*)malloc(N*M * sizeof(float));

	//assign space for obtained result
	float *h_C = (float*)malloc(N*M * sizeof(float));

	//randomly generate a number from the range [0,10] for each cell of matrix
	for (int i = 0; i < N*M; i++) {
		h_A[i] = (float)(rand() % 10 + 1);
		h_B[i] = (float)(rand() % 10 + 1);
	}

	// print the input matrix 
	cout << "Matrix A:" << endl;
	for (int i = 0; i < N*M; i++) {
		cout << h_A[i] << " ";
		if ((i + 1) % N == 0) cout << endl;
	}
	cout << endl;
	cout << "Matrix B:" << endl;
	for (int i = 0; i < N*M; i++) {
		cout << h_B[i] << " ";
		if ((i + 1) % N == 0) cout << endl;
	}
	cout << endl;

	/*
	**matrix product using GPU
	*/

	// create CUBLAS library object and initialise it
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS) {
		
		if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
			cout << "initialisation error of cublas object" << endl;
		}
		getchar();
		return EXIT_FAILURE;
	}

	float *d_A, *d_B, *d_C;
	//allocate matrix space in GPU
	hipMalloc(
		(void**)&d_A, // the head of pointer
		N*M * sizeof(float) //assign the number of Bytes required
	);

	hipMalloc(
		(void**)&d_B,
		N*M * sizeof(float)
	);

	//allocate result matrix in GPU
	hipMalloc(
		(void**)&d_C,
		M*M * sizeof(float)
	);


	hipblasSetVector(
		N*M, //the number of elements 
		sizeof(float), //the size of each element
		h_A, //the start point of GPU address of pointer h_A
		1,	//space break by 1 between two adjacent elements
		d_A, //the start point of GPU address of pointer d_A
		1 //space break by 1 between two adjacent elements
	);

	hipblasSetVector(
		N*M,
		sizeof(float),
		h_B,
		1,
		d_B,
		1
	);

	//synchronization function
	hipDeviceSynchronize();

	//// 传递进矩阵相乘函数中的参数，具体含义请参考函数手册。  
	float a = 1; float b = 0;
	// 矩阵相乘。该函数必然将数组解析成列优先数组  
	hipblasSgemm(
		handle,    // blas 库对象   
		HIPBLAS_OP_T,    // 矩阵 A 属性参数  
		HIPBLAS_OP_T,    // 矩阵 B 属性参数  
		M,    // A, C 的行数   
		M,    // B, C 的列数  
		N,    // A 的列数和 B 的行数  
		&a,    // 运算式的 α 值  
		d_A,    // A 在显存中的地址  
		N,    // lda  
		d_B,    // B 在显存中的地址  
		M,    // ldb  
		&b,    // 运算式的 β 值  
		d_C,    // C 在显存中的地址(结果矩阵)  
		M    // ldc  
	);
	// 同步函数  
	hipDeviceSynchronize();

	// 从 显存 中取出运算结果至 内存中去  
	hipblasGetVector(
		M*M,    //  要取出元素的个数  
		sizeof(float),    // 每个元素大小  
		d_C,    // GPU 端起始地址  
		1,    // 连续元素之间的存储间隔  
		h_C,    // 主机端起始地址  
		1    // 连续元素之间的存储间隔  
	);

	// 打印运算结果  
	cout << "(A*B)^T" << endl;
	for (int i = 0; i < M*M; i++) {
		cout << h_C[i] << " ";
		if ((i + 1) % M == 0) cout << endl;
	}

	// 清理掉使用过的内存  
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// 释放 CUBLAS 库对象  
	hipblasDestroy(handle);

	getchar();

	return 0;
}